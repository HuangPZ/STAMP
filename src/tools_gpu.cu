#include "hip/hip_runtime.h"

#include "tools_gpu.h"

#if (USE_CUDA)
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <stdlib.h> 

#include "tools.h"

// using namespace std;


__global__ void matrixMulGPU(const myType *a, const myType *b, myType *c, size_t N, 
size_t m, size_t l, size_t transpose_a, size_t transpose_b) {
  // Compute each thread's global row and column index
  
	int row = (blockIdx.x * blockDim.x + threadIdx.x)%m;
	int col = (blockIdx.y * blockDim.y + threadIdx.y)%l;
	if (transpose_a){
		if (transpose_b){
			c[row * l + col] = 0;
			for (int k = 0; k < N; k++) {
				// Accumulate results for a single element
				c[row * l + col] += a[2*(row + k*m)] * b[2*(k + col*N)]+
									a[2*(row + k*m)+1] * b[2*(k + col*N)]+
									a[2*(row + k*m)] * b[2*(k + col*N)+1];
			}
		}
		else{
			c[row * l + col] = 0;
			for (int k = 0; k < N; k++) {
				// Accumulate results for a single element
				c[row * l + col] += a[2*(row + k*m)] * b[2*(k * l + col)]+
									a[2*(row + k*m)+1] * b[2*(k * l + col)]+
									a[2*(row + k*m)] * b[2*(k * l + col)+1];
			}
		}
	}
	else{
		if (transpose_b){
			c[row * l + col] = 0;
			for (int k = 0; k < N; k++) {
				// Accumulate results for a single element
				c[row * l + col] += a[2*(row * N + k)] * b[2*(k + col*N)]+
									a[2*(row * N + k)+1] * b[2*(k + col*N)]+
									a[2*(row * N + k)] * b[2*(k + col*N)+1];
			}
		}
		else{
			// Iterate over row, and down column
			c[row * l + col] = 0;
			for (int k = 0; k < N; k++) {
				// Accumulate results for a single element
				c[row * l + col] += a[2*(row * N + k)] * b[2*(k * l + col)]+
									a[2*(row * N + k)+1] * b[2*(k * l + col)]+
									a[2*(row * N + k)] * b[2*(k * l + col)+1];
			}
		}
	}
	
}

void matrixMultRSS_Cuda(const RSSVectorMyType &a, const RSSVectorMyType &b, vector<myType> &temp3, 
	size_t rows, size_t common_dim, size_t columns,size_t transpose_a, size_t transpose_b)
{

	// std::cout<<"!!!!!!!";
	myType *d_a, *d_b, *d_c;
	int myType_size = 4;
	int bytes = 1024;
	// std::cout<<myType_size*2*rows*common_dim<<std::endl;
	int size_a = myType_size*2*rows*common_dim;
	int size_b = myType_size*2*columns*common_dim;
	int size_c = myType_size*rows*columns;
	// std::cout<<size_a<<"??????????"<<size_b<<"??????????"<<size_c<<"??????????"<<std::endl;
	// std::cout<<rows<<"??????????"<<common_dim<<"??????????"<<columns<<"??????????"<<std::endl;
	hipMalloc(&d_a, size_a);
	hipMalloc(&d_b, size_b);
	hipMalloc(&d_c, size_c);
	// for(int i =0;i<size_a;i++){
	// 	std::cout<<a[i].first<<a[i].second<<std::endl;
	// }
	// myType *d_a, *d_b, *d_c;

	// hipMalloc(&d_a, myType_size*2*rows*common_dim);
	// hipMalloc(&d_b, myType_size*2*columns*common_dim);
	// hipMalloc(&d_c, myType_size*2*rows*columns);

	hipMemcpy(d_a, &a[0], size_a, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b[0], size_b, hipMemcpyHostToDevice);
	int THREADS = 32;
	dim3 threads(min(THREADS,(int)rows), min(THREADS,(int)columns));
	dim3 blocks((rows+THREADS-1)/THREADS, (columns+THREADS-1)/THREADS);
	matrixMulGPU<<<blocks, threads>>>(d_a, d_b, d_c, common_dim,rows,columns,transpose_a,transpose_b);

	hipMemcpy(temp3.data(), d_c, size_c, hipMemcpyDeviceToHost);


	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	// check multiplication correct
	// vector<myType> temp_send(rows*columns);
	// matrixMultRSS(a,b, temp_send, 
	// 	 rows,  common_dim,  columns,
	// 	  transpose_a,  transpose_b);
	// for(int i=0;i<columns*rows;i++){
	// 	// std::cout<<i<<" "<<temp_send[i]<<" "<<temp3[i]<<std::endl;
	// 	assert(temp_send[i] == temp3[i]);
	// }

}
__global__ void vectorMulGPU(const myType *a, myType *c, size_t rows, size_t columns ) {
	  // Compute each thread's global row and column index
	  
		int i = (blockIdx.x * blockDim.x + threadIdx.x)%rows;
		int j = (blockIdx.y * blockDim.y + threadIdx.y)%columns;
		
		for (int k = j; k < columns; ++k)
		{
			c[i*columns*columns + j*columns+k] = a[2*(i*columns + k)] * a[2*(i*columns + j)] +
									a[2*(i*columns + k)+1] * a[2*(i*columns + j)] +
									a[2*(i*columns + k)] * a[2*(i*columns + j)+1];
			c[i*columns*columns + j+k*columns] = c[i*columns*columns + j*columns+k];
		}
			

		
	}




void vectorMultRSS_Cuda(const RSSVectorMyType &a, vector<myType> &temp3, size_t rows, size_t columns)		
{
	myType *d_a, *d_b, *d_c;
	int myType_size = 4;
	int bytes = 1024;
	// std::cout<<myType_size*2*rows*common_dim<<std::endl;
	int size_a = myType_size*2*rows*columns;
	int size_c = myType_size*rows*columns*columns;
	// std::cout<<size_a<<"??????????"<<size_b<<"??????????"<<size_c<<"??????????"<<std::endl;
	// std::cout<<rows<<"??????????"<<common_dim<<"??????????"<<columns<<"??????????"<<std::endl;
	hipMalloc(&d_a, size_a);
	hipMalloc(&d_c, size_c);


	hipMemcpy(d_a, &a[0], size_a, hipMemcpyHostToDevice);

	int THREADS = 32;
	dim3 threads(min(THREADS,(int)rows), min(THREADS,(int)columns));
	dim3 blocks((rows+THREADS-1)/THREADS, (columns+THREADS-1)/THREADS);
	vectorMulGPU<<<blocks, threads>>>(d_a, d_c, rows,columns);

	hipMemcpy(temp3.data(), d_c, size_c, hipMemcpyDeviceToHost);


	hipFree(d_a);

	hipFree(d_c);



}

#endif